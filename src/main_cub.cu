#include "image.hh"
#include "pipeline.hh"

#include <vector>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <filesystem>
#include <numeric>

#include <hipcub/hipcub.hpp>

inline void checkCudaCall(hipError_t error, const char* file, int line)
{
    if (error)
    {
        std::cout << "CUDA error at " << file << ":" << line << std::endl;
        std::cout << hipGetErrorName(error) << " :: " << hipGetErrorString(error) << std::endl;
    }
}
#define CHECK_CUDA_CALL(err) (checkCudaCall(err, __FILE__, __LINE__))

struct DifferentFrom
{
    int compare;

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    explicit DifferentFrom(int compare) : compare(compare) {}

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    bool operator()(const int &a) const
    {
        return (a != compare);
    }
};

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    using recursive_directory_iterator = std::filesystem::recursive_directory_iterator;
    std::vector<std::string> filepaths;
    for (const auto& dir_entry : recursive_directory_iterator("./images"))
        filepaths.emplace_back(dir_entry.path()); 

    // - Init pipeline object

    Pipeline pipeline(filepaths);

    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image

    std::cout << "Done, starting compute" << std::endl;

    // #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        // TODO : make it GPU compatible (aka faster)
        // You will need to copy images one by one on the GPU
        // You can store the images the way you want on the GPU
        // But you should treat the pipeline as a pipeline :
        // You *must not* copy all the images and only then do the computations
        // You must get the image from the pipeline as they arrive and launch computations right away
        // There are still ways to speeds this process of course (wait for last class)
        images[i] = pipeline.get_image(i);

        size_t width = images[i].width;
        size_t height = images[i].height;
        int* buffer = &images[i].buffer[0];

        int num_items = images[i].buffer.size();

        // std::cout << "width: " << width <<
        //     " height: " << height <<
        //     " num_items: " << num_items << std::endl;

        int *d_in = NULL;
        CHECK_CUDA_CALL(hipMalloc(&d_in, num_items * sizeof(int)));
        CHECK_CUDA_CALL(hipMemcpy(d_in, buffer, num_items * sizeof(int),
                        hipMemcpyHostToDevice));

        int *d_out = NULL;
        CHECK_CUDA_CALL(hipMalloc(&d_out, num_items * sizeof(int)));

        int *d_num_selected_out = NULL;
        CHECK_CUDA_CALL(hipMalloc(&d_num_selected_out, 1 * sizeof(int)));
        CHECK_CUDA_CALL(hipMemset(d_num_selected_out, 0, 1 * sizeof(int)));
        DifferentFrom select(-27);

        // Determine temporary device storage requirements
        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;

        hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out,
                d_num_selected_out, num_items, select);
        // Allocate temporary storage
        CHECK_CUDA_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));

        // std::cout << "&temp_storage_bytes: " << &temp_storage_bytes << std::endl;
        // std::cout << "temp_storage_bytes: " << temp_storage_bytes << std::endl;

        // Run selection
        hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out,
                d_num_selected_out, num_items, select);

        int d_num_selected_out_host = 0;
        CHECK_CUDA_CALL(hipMemcpy(&d_num_selected_out_host,
                    d_num_selected_out, 1 * sizeof(int), hipMemcpyDeviceToHost));

        // std::cout << "from " << num_items << " to "
        //     << d_num_selected_out_host << std::endl;
        // std::cout << "expected: " << width * height << std::endl;

        assert(d_num_selected_out_host == width * height);

        CHECK_CUDA_CALL(hipFree(d_temp_storage));
        CHECK_CUDA_CALL(hipFree(d_in));
        CHECK_CUDA_CALL(hipFree(d_out));
        CHECK_CUDA_CALL(hipFree(d_num_selected_out));

        return 0;
    }

    std::cout << "Done with compute, starting stats" << std::endl;

    // -- All images are now fixed : compute stats (total then sort)

    // - First compute the total of each image

    // TODO : make it GPU compatible (aka faster)
    // You can use multiple CPU threads for your GPU version using openmp or not
    // Up to you :)
    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        auto& image = images[i];
        const int image_size = image.width * image.height;
        image.to_sort.total = std::reduce(image.buffer.cbegin(), image.buffer.cbegin() + image_size, 0);
    }

    // - All totals are known, sort images accordingly (OPTIONAL)
    // Moving the actual images is too expensive, sort image indices instead
    // Copying to an id array and sort it instead

    // TODO OPTIONAL : for you GPU version you can store it the way you want
    // But just like the CPU version, moving the actual images while sorting will be too slow
    using ToSort = Image::ToSort;
    std::vector<ToSort> to_sort(nb_images);
    std::generate(to_sort.begin(), to_sort.end(), [n = 0, images] () mutable
    {
        return images[n++].to_sort;
    });

    // TODO OPTIONAL : make it GPU compatible (aka faster)
    std::sort(to_sort.begin(), to_sort.end(), [](ToSort a, ToSort b) {
        return a.total < b.total;
    });

    // TODO : Test here that you have the same results
    // You can compare visually and should compare image vectors values and "total" values
    // If you did the sorting, check that the ids are in the same order
    for (int i = 0; i < nb_images; ++i)
    {
        std::cout << "Image #" << images[i].to_sort.id << " total : " << images[i].to_sort.total << std::endl;
        std::ostringstream oss;
        oss << "Image#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
    }

    std::cout << "Done, the internet is safe now :)" << std::endl;

    return 0;
}
