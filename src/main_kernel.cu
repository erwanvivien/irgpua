#include "hip/hip_runtime.h"
#include "image.hh"
#include "pipeline.hh"

#include <vector>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <filesystem>
#include <numeric>

#include <hipcub/hipcub.hpp>

#include <thrust/find.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <thrust/transform.h>
#include <thrust/functional.h>

inline void checkCudaCall(hipError_t error, const char* file, int line)
{
    if (error)
    {
        std::cout << "CUDA error at " << file << ":" << line << std::endl;
        std::cout << hipGetErrorName(error) << " :: " << hipGetErrorString(error) << std::endl;
    }
}
#define CHECK_CUDA_CALL(err) (checkCudaCall(err, __FILE__, __LINE__))

struct DifferentFrom
{
    int compare;

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    explicit DifferentFrom(int compare) : compare(compare) {}

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    bool operator()(const int &a) const
    {
        return (a != compare);
    }
};

struct ToneMap
{
    int *min_histo;
    int *histo;
    int img_dim;

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    explicit ToneMap(int *min_histo, int *histo, int img_dim) : min_histo(min_histo), histo(histo), img_dim(img_dim) {}

    __host__ __device__ __forceinline__
    int operator()(const int &a) const {
        return std::roundf(((histo[a] - *min_histo) / static_cast<float>(img_dim - *min_histo)) * 255.0f);
    }
};

__global__
void cleanup_garbage(int *buffer, int size)
{
    const int tid = threadIdx.x;
    const int coords = tid + blockIdx.x * blockDim.x;

    constexpr const int offset[4] = { 1, -5, 3, -8 };
    buffer[coords] += offset[tid & 0b11]; // mod 4
}

enum State {
    NoCompute = 0,
    SelfCompute = 1,
    AllCompute = 2,
};

template <int STEP>
__device__ void warp_scan(int *internal_buffer_1, int *internal_buffer_2, int tid)
{
    constexpr int left = 1 << STEP;
    if ((tid & left) != 0)
    {
        int right = tid >> STEP;
        int from = left * right - 1;

        internal_buffer_1[tid] += internal_buffer_1[from];
        internal_buffer_2[tid] += internal_buffer_2[from];
    }

    __syncthreads();
}

template <typename T, int BLOCK_SIZE, bool IS_INCLUSIVE = false>
__global__ void sum_scan(T* buffer, int size, int *counter, int* status, int *internal_sum, int* preceeding_sum)
{
    __shared__ T internal_buffer_1[BLOCK_SIZE];
    __shared__ T internal_buffer_2[BLOCK_SIZE];
    __shared__ int blockIdx_x;

    int tid = threadIdx.x;
    if (tid == 0)
        blockIdx_x = atomicAdd(counter, 1);

    __syncthreads();

    int coord = tid + blockIdx_x * (BLOCK_SIZE << 1);
    int value_1 = internal_buffer_1[tid] = buffer[coord];
    int value_2 = internal_buffer_2[tid] = buffer[coord + BLOCK_SIZE];

    __syncthreads();

    // Cumulative sum
    warp_scan<0>(internal_buffer_1, internal_buffer_2, tid); __syncwarp();
    warp_scan<1>(internal_buffer_1, internal_buffer_2, tid); __syncwarp();
    warp_scan<2>(internal_buffer_1, internal_buffer_2, tid); __syncwarp();
    warp_scan<3>(internal_buffer_1, internal_buffer_2, tid); __syncwarp();
    warp_scan<4>(internal_buffer_1, internal_buffer_2, tid); __syncwarp();

    if constexpr (32 < BLOCK_SIZE)
    {
        warp_scan<5>(internal_buffer_1, internal_buffer_2, tid);
        __syncthreads();
    }
    if constexpr (64 < BLOCK_SIZE)
    {
        warp_scan<6>(internal_buffer_1, internal_buffer_2, tid);
        __syncthreads();
    }
    if constexpr (128 < BLOCK_SIZE)
    {
        warp_scan<7>(internal_buffer_1, internal_buffer_2, tid);
        __syncthreads();
    }
    if constexpr (256 < BLOCK_SIZE)
    {
        warp_scan<8>(internal_buffer_1, internal_buffer_2, tid);
        __syncthreads();
    }
    if constexpr (512 < BLOCK_SIZE)
    {
        warp_scan<9>(internal_buffer_1, internal_buffer_2, tid);
        __syncthreads();
    }

    constexpr const int last = BLOCK_SIZE - 1;
    internal_buffer_2[tid] += internal_buffer_1[last];

    int *prefix_sum = preceeding_sum + blockIdx_x;
    int *curr_sum = internal_sum + blockIdx_x;
    int *curr_status = status + blockIdx_x;

    __shared__ int prev_value;

    if (tid == last)
    {
        int local_prev_value = 0;

        atomicExch(curr_sum, internal_buffer_2[last]);
        __threadfence_system();
        atomicExch(curr_status, SelfCompute);

        if (blockIdx_x != 0)
        {
            int back = 1;
            while (back <= blockIdx_x)
            {
                int back_status = atomicAdd(curr_status - back, 0);
                if (back_status == NoCompute)
                {
                    continue;
                }

                else if (back_status == SelfCompute) {
                    local_prev_value += atomicAdd(curr_sum - back, 0);
                    back += 1;
                } else {
                    local_prev_value += atomicAdd(prefix_sum - back, 0);
                    break;
                }
            }
        }

        prev_value = local_prev_value;
    }

    __syncthreads();

    internal_buffer_1[tid] += prev_value;
    internal_buffer_2[tid] += prev_value;

    if (tid == last)
    {
        atomicExch(prefix_sum, internal_buffer_2[last]);
        __threadfence_system();
        atomicExch(curr_status, AllCompute);
    }

    if constexpr (IS_INCLUSIVE)
    {
        buffer[coord] = internal_buffer_1[tid];
        buffer[coord + BLOCK_SIZE] = internal_buffer_2[tid];
    }
    else
    {
        buffer[coord] = internal_buffer_1[tid] - value_1;
        buffer[coord + BLOCK_SIZE] = internal_buffer_2[tid] - value_2;
    }
}

constexpr const long unsigned int expected_total[] = {
    27805567, 185010925, 342970490, 33055988, 390348481,
    91297791, 10825197, 118842538, 72434629, 191735142,
    182802772, 78632198, 491605096, 8109782, 111786760,
    406461934, 80671811, 70004942, 104275727, 30603818,
    6496225, 207334021, 268424419, 432916359, 51973720,
    24489209, 80124196, 29256842, 25803206, 34550754,
};

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    using recursive_directory_iterator = std::filesystem::recursive_directory_iterator;
    std::vector<std::string> filepaths;
    for (const auto& dir_entry : recursive_directory_iterator("./images"))
        filepaths.emplace_back(dir_entry.path()); 

    // - Init pipeline object

    Pipeline pipeline(filepaths);

    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image

    std::cout << "Done, starting compute" << std::endl;

    /// Prepare streams
    constexpr int STREAM_COUNT = 4;
    hipStream_t streams[STREAM_COUNT] = { 0 };
    for (int i = 0; i < STREAM_COUNT; i++)
    {
        hipStreamCreate(streams + i);
    }
    

    // #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        // TODO : make it GPU compatible (aka faster)
        // You will need to copy images one by one on the GPU
        // You can store the images the way you want on the GPU
        // But you should treat the pipeline as a pipeline :
        // You *must not* copy all the images and only then do the computations
        // You must get the image from the pipeline as they arrive and launch computations right away
        // There are still ways to speeds this process of course (wait for last class)
        images[i] = pipeline.get_image(i);


        /// Retrieve image information
        size_t width = images[i].width;
        size_t height = images[i].height;
        int* buffer = &images[i].buffer[0];
        int num_items = images[i].buffer.size();

        int img_dim = width * height;

        constexpr int blocksize = 1024;
        const int gridsize = (img_dim + blocksize - 1) / blocksize;

        /// Retrieve the attached stream
        hipStream_t stream = streams[i % 4];

        /// Prepare CUDA buffer (image input)
        int *d_in = NULL;
        CHECK_CUDA_CALL(hipMallocAsync(&d_in, num_items * sizeof(int), stream));
        CHECK_CUDA_CALL(hipMemcpyAsync(d_in, buffer, num_items * sizeof(int),
                        hipMemcpyHostToDevice, stream));

        /// Prepare CUDA buffer (image without -27s)
        int *d_out = NULL;
        CHECK_CUDA_CALL(hipMallocAsync(&d_out, num_items * sizeof(int), stream));

        /// Retrieve the information
        int *d_num_selected_out = NULL;
        CHECK_CUDA_CALL(hipMallocAsync(&d_num_selected_out, 1 * sizeof(int), stream));
        CHECK_CUDA_CALL(hipMemsetAsync(d_num_selected_out, 0, 1 * sizeof(int), stream));

        /// Create a class with overloaded bool operator
        DifferentFrom select(-27);

        // Determine temporary device storage requirements
        {
            void     *d_temp_storage = NULL;
            size_t   temp_storage_bytes = 0;
            int d_num_selected_out_host = 0;

            hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out,
                    d_num_selected_out, num_items, select, stream);
            // Allocate temporary storage
            CHECK_CUDA_CALL(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));

            // Run selection (removes -27s)
            hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out,
                    d_num_selected_out, num_items, select, stream);

            /// Move to CPU side the count of item to check
            /// TODO: Remove this (useless)
            CHECK_CUDA_CALL(hipMemcpyAsync(&d_num_selected_out_host,
                        d_num_selected_out, 1 * sizeof(int), hipMemcpyDeviceToHost, stream));

            assert(d_num_selected_out_host == img_dim);

            CHECK_CUDA_CALL(hipFreeAsync(d_temp_storage, stream));
        }

        /// Remove the random garbage values from the array
        {
            cleanup_garbage<<<gridsize, blocksize, 0, stream>>>(d_out, img_dim);
        }

        /// Compute histogram
        int*     d_histogram = NULL;
        CHECK_CUDA_CALL(hipMallocAsync(&d_histogram, 256 * sizeof(int), stream));
        CHECK_CUDA_CALL(hipMemsetAsync(d_histogram, 0, 256 * sizeof(int), stream));

        {
            void*    d_temp_storage = NULL;
            size_t   temp_storage_bytes = 0;

            int      num_samples = img_dim;
            int*   d_samples = d_out;
            int num_levels  = 256 + 1;
            int lower_level = 0;
            int upper_level = 256;

            hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes,
                    d_samples, d_histogram, num_levels, lower_level, upper_level, num_samples, stream);

            // Allocate temporary storage
            CHECK_CUDA_CALL(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
            // Compute histograms
            hipcub::DeviceHistogram::HistogramEven(d_temp_storage, temp_storage_bytes,
                    d_samples, d_histogram, num_levels, lower_level, upper_level, num_samples, stream);

            CHECK_CUDA_CALL(hipFreeAsync(d_temp_storage, stream));
        }

        /// Replace histogram with cumulative histogram
        {
            constexpr const int blocksize = 128;
            const int gridsize = (256 + blocksize - 1) / (blocksize * 2);

            int *counter = NULL;
            CHECK_CUDA_CALL(hipMalloc(&counter, 1 * sizeof(int)));
            CHECK_CUDA_CALL(hipMemset(counter, 0, 1 * sizeof(int)));

            int *status = NULL;
            CHECK_CUDA_CALL(hipMalloc(&status, gridsize * sizeof(int)));
            CHECK_CUDA_CALL(hipMemset(status, NoCompute, gridsize * sizeof(int)));

            int *internal_sum = NULL;
            CHECK_CUDA_CALL(hipMalloc(&internal_sum, gridsize * sizeof(int)));
            CHECK_CUDA_CALL(hipMemset(internal_sum, 0, gridsize * sizeof(int)));

            int *preceeding_sum = NULL;
            CHECK_CUDA_CALL(hipMalloc(&preceeding_sum, gridsize * sizeof(int)));
            CHECK_CUDA_CALL(hipMemset(preceeding_sum, 0, gridsize * sizeof(int)));

            sum_scan<int, blocksize, true><<<gridsize, blocksize, 0, stream>>>(d_histogram, 256, counter, status, internal_sum, preceeding_sum);

            CHECK_CUDA_CALL(hipFree(counter));
            CHECK_CUDA_CALL(hipFree(status));
            CHECK_CUDA_CALL(hipFree(internal_sum));
            CHECK_CUDA_CALL(hipFree(preceeding_sum));
        }

        /// Apply histogram equalization
        {
            auto policy = thrust::cuda::par.on(stream);

            auto iter = thrust::find_if(policy, d_histogram, d_histogram + 256, DifferentFrom(0));
            ToneMap tonemap(iter, d_histogram, img_dim);
            thrust::transform(policy, d_out, d_out + img_dim, d_out, tonemap);
        }

        /// Compute reduce
        int *reduce_sum = NULL;
        CHECK_CUDA_CALL(hipMallocAsync(&reduce_sum, 1 * sizeof(int), stream));
        CHECK_CUDA_CALL(hipMemsetAsync(reduce_sum, 0, 1 * sizeof(int), stream));

        // - First compute the total of each image

        // TODO : make it GPU compatible (aka faster)
        // You can use multiple CPU threads for your GPU version using openmp or not
        // Up to you :)
        {
            void     *d_temp_storage = NULL;
            size_t   temp_storage_bytes = 0;

            hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_out, reduce_sum, img_dim, stream);
            // Allocate temporary storage
            CHECK_CUDA_CALL(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));
            // Run sum-reduction
            hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_out, reduce_sum, img_dim, stream);

            CHECK_CUDA_CALL(hipFreeAsync(d_temp_storage, stream));
        }

        /// Not mandatory (we are not using values past img_dim)
        images[i].buffer.resize(img_dim);

        /// Retrieve the image from GPU
        CHECK_CUDA_CALL(hipMemcpyAsync(buffer, d_out, img_dim * sizeof(int), hipMemcpyDeviceToHost, stream));
        /// Retrieve the total from GPU
        CHECK_CUDA_CALL(hipMemcpyAsync(&images[i].to_sort.total, reduce_sum, 1 * sizeof(int), hipMemcpyDeviceToHost, stream));

        /// Clean everything
        CHECK_CUDA_CALL(hipFreeAsync(d_in, stream));
        CHECK_CUDA_CALL(hipFreeAsync(d_out, stream));
        CHECK_CUDA_CALL(hipFreeAsync(d_num_selected_out, stream));
        CHECK_CUDA_CALL(hipFreeAsync(reduce_sum, stream));
        CHECK_CUDA_CALL(hipFreeAsync(d_histogram, stream));
    }

    /// Cleanup streams
    for (int i = 0; i < STREAM_COUNT; i++)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    std::cout << "Done with compute, starting stats" << std::endl;

    // -- All images are now fixed : compute stats (total then sort)


    // - All totals are known, sort images accordingly (OPTIONAL)
    // Moving the actual images is too expensive, sort image indices instead
    // Copying to an id array and sort it instead

    // TODO OPTIONAL : for you GPU version you can store it the way you want
    // But just like the CPU version, moving the actual images while sorting will be too slow
    using ToSort = Image::ToSort;
    std::vector<ToSort> to_sort(nb_images);
    std::generate(to_sort.begin(), to_sort.end(), [n = 0, images] () mutable
    {
        return images[n++].to_sort;
    });

    // TODO OPTIONAL : make it GPU compatible (aka faster)
    std::sort(to_sort.begin(), to_sort.end(), [](ToSort a, ToSort b) {
        return a.total < b.total;
    });

    // TODO : Test here that you have the same results
    // You can compare visually and should compare image vectors values and "total" values
    // If you did the sorting, check that the ids are in the same order
    for (int i = 0; i < nb_images; ++i)
    {
        std::cout << "Image #" << images[i].to_sort.id << " total : " << images[i].to_sort.total << std::endl;
        std::ostringstream oss;
        oss << "Image#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
    }

    for (int i = 0; i < nb_images; ++i) {
        auto &img = images[i];
        if (img.to_sort.total != expected_total[img.to_sort.id]) {
            std::cerr << "Differ computed image " << i << ": (" << img.to_sort.total <<
                ") expected " << expected_total[img.to_sort.id] << std::endl;
        }
    }

    std::cout << "Done, the internet is safe now :)" << std::endl;

    return 0;
}
